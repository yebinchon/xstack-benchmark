/**
 * gemm.c: This file is part of the PolyBench/C 3.2 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

__global__ void kernel_dev(int ni, int nj, int nk,
                           double alpha,
                           double beta,
                           double *C, double *A, double *B) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int k;


  if (i < ni && j < nj) {
    C[i * nj + j] *= beta;


    for (k = 0; k < nk; k++)
      C[i * nj + j] += alpha * A[i * nk + k] * B[k * nj + j];
  }
}


static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}

static void kernel(int ni, int nj, int nk,
                   double *alpha,
                   double *beta,
                   double *C, double *A, double *B) {

  double *dev_A;
  double *dev_B;
  double *dev_C;
  double *dev_alpha;
  double *dev_beta;
  hipMalloc(&dev_A, ni*nk*sizeof(double));
  hipMalloc(&dev_B, nk*nj*sizeof(double));
  hipMalloc(&dev_C, ni*nj*sizeof(double));
  hipMalloc(&dev_alpha, sizeof(double));
  hipMalloc(&dev_beta, sizeof(double));
  hipMemcpy(dev_A, A, ni*nk*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, nk*nj*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_C, C, ni*nj*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_alpha, alpha, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_beta, beta, sizeof(double), hipMemcpyHostToDevice);

  unsigned threadsPerBlock = 256;
  dim3 block(threadsPerBlock / 32, 32, 1);
  dim3 grid(num_blocks(ni, block.x), num_blocks(nj, block.y), 1);
  kernel_dev<<<grid, block>>>(ni, nj, nk, *dev_alpha, *dev_beta, dev_C, dev_A, dev_B);

  hipMemcpy(C, dev_C, ni*nj*sizeof(double), hipMemcpyDeviceToHost);
  hipFree((void*)dev_A);
  hipFree((void*)dev_B);
  hipFree((void*)dev_C);
  hipFree((void*)dev_alpha);
  hipFree((void*)dev_beta);
}

  static
void init_array(int ni, int nj, int nk,
    double *alpha,
    double *beta,
    double *C,
    double *A,
    double *B)
{
  int i, j;

  *alpha = 32412;
  *beta = 2123;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++)
      C[i*nj+j] = ((double) i*j) / ni;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nk; j++)
      A[i*nk+j] = ((double) i*j) / ni;
  for (i = 0; i < nk; i++)
    for (j = 0; j < nj; j++)
      B[i*nj+j] = ((double) i*j) / ni;
}




  static
void print_array(int ni, int nj,
    double *C)
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
      fprintf (stderr, "%0.2lf ", C[i*nj+j]);
      if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


int main(int argc, char** argv)
{

  int dump_code = atoi(argv[1]);
  int ni = atoi(argv[2]);
  int nj = atoi(argv[3]);
  int nk = atoi(argv[4]);

  double *alpha = (double*)malloc(sizeof(double));
  double *beta = (double*)malloc(sizeof(double));
  double *A = (double*)malloc(ni*nk*sizeof(double));
  double *B = (double*)malloc(nk*nj*sizeof(double));
  double *C = (double*)malloc(ni*nj*sizeof(double));


  init_array (ni, nj, nk, alpha, beta,
      C,
      A,
      B);



  kernel(ni, nj, nk, alpha, beta, C, A, B);




  if (dump_code == 1) print_array(ni, nj, C);


  free((void*)C);
  free((void*)A);
  free((void*)B);
  free((void*)alpha);
  free((void*)beta);

  return 0;
}
