/**
 * bicg.c: This file is part of the PolyBench 3.0 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

//#define NX 40000
#define RUN 50

static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}



__global__ void kernel_q(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    q[i] = 0;
    for (int j = 0; j < m; j++)
      q[i] += A[i * m + j] * p[j];
  }
}


__global__ void kernel_s(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if (j < m) {
    s[j] = 0;
    for (int i = 0; i < n; i++)
      s[j] += r[i] * A[i * m + j];
  }
}


/* Array initialization. */
static
void init_array (int nx, int ny,
		 double *A,
		 double *r,
		 double *p)
{
  int i, j;

  for (i = 0; i < ny; i++)
    p[i] = i * M_PI;
  for (i = 0; i < nx; i++) {
    r[i] = i * M_PI;
    for (j = 0; j < ny; j++)
      A[i*ny+j] = ((double) i*(j+1))/nx;
  }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx, int ny,
		 double *s,
		 double *q)

{
  int i;

  for (i = 0; i < ny; i++) {
    fprintf (stderr, "%0.2lf ", s[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  for (i = 0; i < nx; i++) {
    fprintf (stderr, "%0.2lf ", q[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  fprintf (stderr, "\n");
}


static void kernel(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  double *dev_A;
  double *dev_s;
  double *dev_q;
  double *dev_p;
  double *dev_r;
  hipMalloc(&dev_A, m*n*sizeof(double));
  hipMalloc(&dev_s, n*sizeof(double));
  hipMalloc(&dev_q, m*sizeof(double));
  hipMalloc(&dev_p, n*sizeof(double));
  hipMalloc(&dev_r, m*sizeof(double));
  hipMemcpy(dev_A, A, m*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_s, s, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_q, q, m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, r, m*sizeof(double), hipMemcpyHostToDevice);
  const unsigned threadsPerBlock = 256;
  kernel_q<<<num_blocks(n, threadsPerBlock), threadsPerBlock>>>(m, n, dev_A, dev_s, dev_q, dev_p, dev_r);
  kernel_s<<<num_blocks(m, threadsPerBlock), threadsPerBlock>>>(m, n, dev_A, dev_s, dev_q, dev_p, dev_r);
  hipMemcpy(s, dev_s, n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(q, dev_q, m*sizeof(double), hipMemcpyDeviceToHost);
  hipFree((void*)dev_A);
  hipFree((void*)dev_s);
  hipFree((void*)dev_q);
  hipFree((void*)dev_p);
  hipFree((void*)dev_r);
}

int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int nx =atoi(argv[2]);
  int ny = atoi(argv[3]);
  int dump_code = atoi(argv[1]);

  for(int t = 0; t < RUN; t++) {

  /* Variable declaration/allocation. */
  double *A = (double*)malloc(nx*ny*sizeof(double));
  double *s = (double*)malloc(ny*sizeof(double));
  double *q = (double*)malloc(nx*sizeof(double));
  double *p = (double*)malloc(ny*sizeof(double));
  double *r = (double*)malloc(nx*sizeof(double));
  /* Initialize array(s). */
  init_array (nx, ny, A, r, p);


  kernel(nx, ny, A, s, q, p, r);

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  if(dump_code == 1) print_array(nx, ny, s, q);

  /* Be clean. */
  free((void*)A);
  free((void*)s);
  free((void*)q);
  free((void*)p);
  free((void*)r);

  }

  return 0;
}
