/**
 * syr2k.c: This file is part of the PolyBench/C 3.2 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>


static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}


__global__ void kernel_beta(int n, int m,
                            double alpha, double beta,
                            double *C,
                            double *A,
                            double *B) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;


  if (i < n && j <= i)
    C[i * n + j] *= beta;
}

__global__ void kernel_product(int n, int m,
                               double alpha, double beta,
                               double *C,
                               double *A,
                               double *B) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;


  if (i < n && j <= i) {
    for (int k = 0; k < m; k++)
      C[i * n + j] += A[j * m + k] * alpha * B[i * m + k] + B[j * m + k] * alpha * A[i * m + k];
  }
}



static void kernel(int n, int m,
                   double alpha, double beta,
                   double *C,
                   double *A,
                   double *B) {
  const unsigned int threadsPerBlock = 256;

  {
    dim3 block(threadsPerBlock / 32, 32, 1);
    dim3 grid(num_blocks(n, block.x), num_blocks(n, block.y), 1);
    kernel_beta<<<grid, block>>>(n, m, alpha, beta, C, A, B);
  }

  {
    dim3 block(threadsPerBlock / 32, 32, 1);
    dim3 grid(num_blocks(n, block.x), num_blocks(n, block.y), 1);
    kernel_product<<<grid, block>>>(n, m, alpha, beta, C, A, B);
  }

}

  static
void init_array(int ni, int nj,
    double *C,
    double *A,
    double *B)
{
  int i, j;


  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
      A[i*nj+j] = ((double) i*j) / ni;
      B[i*nj+j] = ((double) i*j) / ni;
    }
  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++)
      C[i*ni+j] = ((double) i*j) / ni;
}




  static
void print_array(int ni,
    double *C)
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < ni; j++) {
      fprintf (stderr, "%0.2lf ", C[i*ni+j]);
      if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}





int main(int argc, char** argv)
{
  int dump_code = atoi(argv[1]);
  int m = atoi(argv[2]);
  int n = atoi(argv[3]);

  double *A = (double*)malloc(m*n*sizeof(double));
  double *B = (double*)malloc(m*n*sizeof(double));
  double *C = (double*)malloc(m*m*sizeof(double));

  double alpha = 32412;
  double beta = 2123;

  init_array (m, n,
      C,
      A,
      B);


  double *dev_A;
  double *dev_B;
  double *dev_C;
  hipMalloc(&dev_A, n*m*sizeof(double));
  hipMalloc(&dev_B, n*m*sizeof(double));
  hipMalloc(&dev_C, n*m*sizeof(double));
  hipMemcpy(dev_A, A, n*m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, n*m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_C, C, n*m*sizeof(double), hipMemcpyHostToDevice);



  kernel(m, n, alpha, beta, dev_C, dev_A, dev_B);
  hipMemcpy(C, dev_C, n*m*sizeof(double), hipMemcpyDeviceToHost);




  if (dump_code == 1) print_array(m, C);


  free((void*)C);;
  free((void*)A);;
  free((void*)B);;

  return 0;
}
