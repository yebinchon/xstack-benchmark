#include "hip/hip_runtime.h"
/* Provide Declarations */
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#ifndef __cplusplus
typedef unsigned char bool;
#endif

#ifndef _MSC_VER
#define __forceinline __attribute__((always_inline)) inline
#endif

#if defined(__GNUC__)
#define  __ATTRIBUTELIST__(x) __attribute__(x)
#else
#define  __ATTRIBUTELIST__(x)  
#endif

#ifdef _MSC_VER  /* Can only support "linkonce" vars with GCC */
#define __attribute__(X)
#endif



/* Global Declarations */

/* Types Declarations */
struct l_struct_struct_OC__IO_FILE;

/* Function definitions */

/* Types Definitions */
struct l_array_1_uint8_t {
  uint8_t array[1];
};
struct l_array_20_uint8_t {
  uint8_t array[20];
};
struct l_struct_struct_OC__IO_FILE {
  uint32_t field0;
  uint8_t* field1;
  uint8_t* field2;
  uint8_t* field3;
  uint8_t* field4;
  uint8_t* field5;
  uint8_t* field6;
  uint8_t* field7;
  uint8_t* field8;
  uint8_t* field9;
  uint8_t* field10;
  uint8_t* field11;
  void* field12;
  struct l_struct_struct_OC__IO_FILE* field13;
  uint32_t field14;
  uint32_t field15;
  uint64_t field16;
  uint16_t field17;
  uint8_t field18;
  uint8_t field19[1];
  uint8_t* field20;
  uint64_t field21;
  void* field22;
  void* field23;
  struct l_struct_struct_OC__IO_FILE* field24;
  uint8_t* field25;
  uint64_t field26;
  uint32_t field27;
  uint8_t field28[20];
};

/* External Global Variable Declarations */

/* Function Declarations */
int main(int, char **) __ATTRIBUTELIST__((noinline, nothrow));
void init_array(uint32_t, uint32_t, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void kernel_atax(uint32_t, uint32_t, double*, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void print_array(uint32_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel0(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel1(uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void kernel_atax_OC_cudakernel0(uint64_t, uint64_t, uint64_t, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void kernel_atax_OC_cudakernel1(uint64_t, double*, uint32_t, uint64_t, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void kernel_atax_OC_cudakernel2(uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));


/* Global Variable Definitions and Initialization */
uint8_t _OC_str[8] = { "%0.2lf " };
uint8_t _OC_str_OC_1[2] = { "\n" };


/* LLVM Intrinsic Builtin Function Bodies */
static __forceinline uint64_t llvm_add_u64(uint64_t a, uint64_t b) {
  uint64_t r = a + b;
  return r;
}
static __forceinline uint32_t llvm_mul_u32(uint32_t a, uint32_t b) {
  uint32_t r = a * b;
  return r;
}
static __forceinline uint64_t llvm_mul_u64(uint64_t a, uint64_t b) {
  uint64_t r = a * b;
  return r;
}
static __forceinline uint32_t llvm_urem_u32(uint32_t a, uint32_t b) {
  uint32_t r = a % b;
  return r;
}


/* Function Bodies */

int main(int argc, char ** argv) {
  int32_t nx;
  int32_t ny;
  int32_t dump_code;
  uint8_t* A;
  uint8_t* x;
  uint8_t* y;
  uint8_t* tmp;

  nx = atoi(argv[2]);
  ny = atoi(argv[3]);
  dump_code = atoi(argv[1]);
  A = malloc(nx * ny * 8);
  x = malloc(ny * 8);
  y = malloc(nx * 8);
  tmp = malloc(ny * 8);
  init_array(nx, ny, ((double*)A), ((double*)x));
;
  kernel_atax(nx, ny, ((double*)A), ((double*)x), ((double*)y), ((double*)tmp));
;
  if (dump_code == 1) {
print_array(nx, ((double*)y));
  }
free(((uint8_t*)((double*)A)));
free(((uint8_t*)((double*)x)));
free(((uint8_t*)((double*)y)));
free(((uint8_t*)((double*)tmp)));
  return 0;
}


void init_array(uint32_t nx, uint32_t ny, double* A, double* x) {
init_array_OC_cudakernel1<<<grid, block>>>(ny, x);
init_array_OC_cudakernel0<<<grid, block>>>(nx, ny, nx, ny, A);
  return;
}


void kernel_atax(uint32_t nx, uint32_t ny, double* A, double* x, double* y, double* tmp) {
kernel_atax_OC_cudakernel2<<<grid, block>>>(nx, y);
kernel_atax_OC_cudakernel1<<<grid, block>>>(ny, tmp, ny, ny, A, x);
kernel_atax_OC_cudakernel0<<<grid, block>>>(nx, ny, ny, A, tmp, y);
  return;
}


void print_array(uint32_t nx, double* y) {
  int64_t i;
  int32_t call3;


for(int64_t i = 0; i < nx;   i = i + 1){
  uint32_t call = fprintf(stderr, _OC_str, y[i]);
  if (i % 20 == 0) {
  fprintf(stderr, _OC_str_OC_1);
  }
}
  fprintf(stderr, _OC_str_OC_1);
}


__global__ void init_array_OC_cudakernel0(uint64_t _1, uint64_t _2, uint32_t nx, uint64_t _3, double* A) {
  int64_t indvars_2e_iv1;
  int64_t i;
  uint64_t indvars_2e_iv_2e_next2;

  indvars_2e_iv1 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (indvars_2e_iv1 < _1) {

for(int64_t i = 0; i < _2;   i = i + 1){
  (A+indvars_2e_iv1 * _3)[i] = (((double)(indvars_2e_iv1) * (double)((i + 1))) / (double)(nx));
}
  indvars_2e_iv1 + 1;
  }
  return;
}


__global__ void init_array_OC_cudakernel1(uint64_t _4, double* x) {
  int64_t indvars_2e_iv3;
  uint64_t indvars_2e_iv_2e_next4;

  indvars_2e_iv3 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (indvars_2e_iv3 < _4) {
  x[indvars_2e_iv3] = ((double)(indvars_2e_iv3) * 3.1415926535897931);
  indvars_2e_iv3 + 1;
  }
  return;
}


__global__ void kernel_atax_OC_cudakernel0(uint64_t _5, uint64_t _6, uint64_t _7, double* A, double* tmp, double* y) {
  int64_t indvars_2e_iv1;
  int64_t i;
  uint64_t indvars_2e_iv_2e_next2;

  indvars_2e_iv1 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (indvars_2e_iv1 < _5) {

for(int64_t i = 0; i < _6;   i = i + 1){
  y[indvars_2e_iv1] = (y[indvars_2e_iv1] + ((A+i * _7)[indvars_2e_iv1] * tmp[i]));
}
  indvars_2e_iv1 + 1;
  }
  return;
}


__global__ void kernel_atax_OC_cudakernel1(uint64_t _8, double* tmp, uint32_t ny, uint64_t _9, double* A, double* x) {
  int64_t indvars_2e_iv5;
  int64_t i;
  uint64_t indvars_2e_iv_2e_next6;

  indvars_2e_iv5 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (indvars_2e_iv5 < _8) {
  tmp[indvars_2e_iv5] = 0;

for(int64_t i = 0; i < ny;   i = i + 1){
  tmp[indvars_2e_iv5] = (tmp[indvars_2e_iv5] + ((A+indvars_2e_iv5 * _9)[i] * x[i]));
}
  indvars_2e_iv5 + 1;
  }
  return;
}


__global__ void kernel_atax_OC_cudakernel2(uint64_t _10, double* y) {
  int64_t indvars_2e_iv7;
  uint64_t indvars_2e_iv_2e_next8;

  indvars_2e_iv7 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (indvars_2e_iv7 < _10) {
  y[indvars_2e_iv7] = 0;
  indvars_2e_iv7 + 1;
  }
  return;
}

