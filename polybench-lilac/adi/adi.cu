/**
 * adi.c: This file is part of the PolyBench 3.0 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

#define N 2048
#define TSTEPS 50


__global__ void kernel_column_sweep(int tsteps,
                                    int n,
                                    double *u,
                                    double *v,
                                    double *p,
                                    double *q, double a, double b, double c, double d, double e, double f) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;


  if (i < n - 1) {
    v[0 * n + i] = 1;
    p[i * n + 0] = 0;
    q[i * n + 0] = v[0 * n + i];
    for (int j = 1; j < n - 1; j++) {
      p[i * n + j] = -c / (a * p[i * n + j - 1] + b);
      q[i * n + j] = (-d * u[j * n + i - 1] + (1 + 2 * d) * u[j * n + i] - f * u[j * n + i + 1] - a * q[i * n + j - 1]) / (a * p[i * n + j - 1] + b);
    }

    v[(n - 1) * n + i] = 1;
    for (int j = n - 2; j >= 1; j--)
      v[j * n + i] = p[i * n + j] * v[(j + 1) * n + i] + q[i * n + j];
  }
}


__global__ void kernel_row_sweep(int tsteps, int n, double *u, double *v, double *p, double *q, double a, double b, double c, double d, double e, double f) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;

  if (i < n - 1) {
    u[i * n + 0] = 1;
    p[i + n + 0] = 0;
    q[i * n + 0] = u[i * n + 0];
    for (int j = 1; j < n - 1; j++) {
      p[i * n + j] = -f / (d * p[i * n + j - 1] + e);
      q[i * n + j] = (-a * v[(i - 1) * n + j] + (1 + 2 * a) * v[i * n + j] - c * v[(i + 1) * n + j] - d * q[i * n + j - 1]) / (d * p[i * n + j - 1] + e);
    }
    u[i * n + n - 1] = 1;
    for (int j =n - 2; j >= 1; j--)
      u[i * n + j] = p[i * n + j] * u[i * n + j + 1] + q[i * n + j];
  }
}

static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}



static void kernel(
    int tsteps,
    int n,
    double *u,
    double *v,
    double *p,
    double *q) {
  unsigned threadsPerBlock = 256;

  double DX = 1 / (double)n;
  double DY = 1 / (double)n;
  double DT = 1 / (double)tsteps;
  double B1 = 2;
  double B2 = 1;
  double mul1 = B1 * DT / DX / DX;
  double mul2 = B2 * DT / DY / DY;

  double a = -mul1 / 2;
  double b = 1 + mul1;
  double c = a;
  double d = -mul2 / 2;
  double e = 1 + mul2;
  double f = d;




  for (int t = 1; t <= tsteps; t++) {
    // Column Sweep
    kernel_column_sweep<<<num_blocks(n - 2, threadsPerBlock), threadsPerBlock>>>(tsteps, n, u, v, p, q, a, b, c, d, e, f);

    // Row Sweep
    kernel_row_sweep<<<num_blocks(n - 2, threadsPerBlock), threadsPerBlock>>>(tsteps, n, u, v, p, q, a, b, c, d, e, f);
  }

}

/* Array initialization. */
static void init_array (int n,
		 double *u, double *v, double*p, double*q)
{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++){
	    u[i*n+j] =  (double)(i + n-j) / n;
      v[i*n+j] = 0;
      p[i*n+j] = 0;
      q[i*n+j] = 0;
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 double *u)

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      fprintf(stderr, "%0.2lf ", u[i*n+j]);
      if ((i * n + j) % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = atoi(argv[2]);
  int tsteps = atoi(argv[3]);
  int dump_code = atoi(argv[1]);

  /* Variable declaration/allocation. */

  double *u = (double*)malloc(n*n*sizeof(double));
  double *v = (double*)malloc(n*n*sizeof(double));
  double *p = (double*)malloc(n*n*sizeof(double));
  double *q = (double*)malloc(n*n*sizeof(double));


  /* Initialize array(s). */
  init_array (n, u, v, p, q);


  double *dev_u;
  double *dev_v;
  double *dev_p;
  double *dev_q;
  hipMalloc(&dev_u, n*n*sizeof(double));
  hipMalloc(&dev_v, n*n*sizeof(double));
  hipMalloc(&dev_p, n*n*sizeof(double));
  hipMalloc(&dev_q, n*n*sizeof(double));
  hipMemcpy(dev_u, u, n*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_v, v, n*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p, n*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_q, q, n*n*sizeof(double), hipMemcpyHostToDevice);

  /* Run kernel. */
  kernel(tsteps, n, dev_u, dev_v, dev_p, dev_q);

  hipMemcpy(u, dev_u, n*n*sizeof(double), hipMemcpyDeviceToHost);
  hipFree((void*)dev_u);
  hipFree((void*)dev_v);
  hipFree((void*)dev_p);
  hipFree((void*)dev_q);

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  if(dump_code == 1) print_array(n, u);

  /* Be clean. */
  free((void*)u);
  free((void*)v);
  free((void*)p);
  free((void*)q);


  return 0;
}
