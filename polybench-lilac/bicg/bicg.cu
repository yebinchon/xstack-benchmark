/**
 * bicg.c: This file is part of the PolyBench 3.0 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

//#define NX 40000
#define RUN 100

static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}



__global__ void kernel_q(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n) {
    double dot = 0;
    q[i] = 0;
    for (int j = 0; j < m; j++)
      dot += A[i * m + j] * p[j];
    q[i] += dot;
  }
}


__global__ void kernel_s(int m, int n, double *A, double s[], double q[], double p[], double r[]) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if (j < m) {
    s[j] = 0;
    double dot = 0;
    for (int i = 0; i < n; i++)
      dot += r[i] * A[i * m + j];
    s[j]= dot;
  }
}


/* Array initialization. */
static
void init_array (int nx, int ny,
		 double *A,
		 double *r,
		 double *p)
{
  int i, j;

  for (i = 0; i < ny; i++)
    p[i] = i * M_PI;
  for (i = 0; i < nx; i++) {
    r[i] = i * M_PI;
    for (j = 0; j < ny; j++)
      A[i*ny+j] = ((double) i*(j+1))/nx;
  }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx, int ny,
		 double *s,
		 double *q)

{
  int i;

  for (i = 0; i < ny; i++) {
    fprintf (stderr, "%0.2lf ", s[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  for (i = 0; i < nx; i++) {
    fprintf (stderr, "%0.2lf ", q[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  fprintf (stderr, "\n");
}


static void kernel(int m, int n, double *A, double s[], double q[], double p[], double r[]) {

  const unsigned threadsPerBlock = 256;
  kernel_q<<<num_blocks(n, threadsPerBlock), threadsPerBlock>>>(m, n, A, s, q, p, r);
  kernel_s<<<num_blocks(m, threadsPerBlock), threadsPerBlock>>>(m, n, A, s, q, p, r);

}

int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int m =atoi(argv[2]);
  int n = atoi(argv[3]);
  int dump_code = atoi(argv[1]);

//  for(int i = 0; i < RUN; i++) {
  /* Variable declaration/allocation. */
  double *A = (double*)malloc(m*n*sizeof(double));
  double *s = (double*)malloc(n*sizeof(double));
  double *q = (double*)malloc(m*sizeof(double));
  double *p = (double*)malloc(n*sizeof(double));
  double *r = (double*)malloc(m*sizeof(double));
  /* Initialize array(s). */
  init_array (m,n, A, r, p);
  double *dev_A;
  double *dev_s;
  double *dev_q;
  double *dev_p;
  double *dev_r;
  hipMalloc(&dev_A, m*n*sizeof(double));
  hipMalloc(&dev_s, n*sizeof(double));
  hipMalloc(&dev_q, m*sizeof(double));
  hipMalloc(&dev_p, n*sizeof(double));
  hipMalloc(&dev_r, m*sizeof(double));
  hipMemcpy(dev_A, A, m*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_s, s, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_q, q, m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p, n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, r, m*sizeof(double), hipMemcpyHostToDevice);
  kernel(m,n, dev_A, dev_s, dev_q, dev_p, dev_r);
  hipMemcpy(s, dev_s, n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(q, dev_q, m*sizeof(double), hipMemcpyDeviceToHost);
  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  if(dump_code == 1) print_array(m, n, s, q);
  /* Be clean. */
  free((void*)A);
  free((void*)s);
  free((void*)q);
  free((void*)p);
  free((void*)r);
//  }

  return 0;
}
