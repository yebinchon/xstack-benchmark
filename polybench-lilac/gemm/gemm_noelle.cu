#include "hip/hip_runtime.h"
/* Provide Declarations */
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#ifndef __cplusplus
typedef unsigned char bool;
#endif

#ifndef _MSC_VER
#define __forceinline __attribute__((always_inline)) inline
#endif

#if defined(__GNUC__)
#define  __ATTRIBUTELIST__(x) __attribute__(x)
#else
#define  __ATTRIBUTELIST__(x)  
#endif

#ifdef _MSC_VER  /* Can only support "linkonce" vars with GCC */
#define __attribute__(X)
#endif



/* Global Declarations */

/* Types Declarations */
struct __FIXME__l_struct_struct_OC__IO_FILE;

/* Function definitions */

/* Types Definitions */
struct __FIXME__l_array_1_uint8_t {
  uint8_t array[1];
};
struct __FIXME__l_array_20_uint8_t {
  uint8_t array[20];
};
struct __FIXME__l_struct_struct_OC__IO_FILE {
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field0;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field1;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field2;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field3;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field4;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field5;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field6;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field7;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field8;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field9;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field10;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field11;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field12;
  struct __FIXME__l_struct_struct_OC__IO_FILE* __FIXME__l_struct_struct_OC__IO_FILE_field13;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field14;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field15;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field16;
  uint16_t __FIXME__l_struct_struct_OC__IO_FILE_field17;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field18;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field19[1];
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field20;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field21;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field22;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field23;
  struct __FIXME__l_struct_struct_OC__IO_FILE* __FIXME__l_struct_struct_OC__IO_FILE_field24;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field25;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field26;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field27;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field28[20];
};

/* External Global Variable Declarations */

/* Function Declarations */
int main(int, char **) __ATTRIBUTELIST__((noinline, nothrow));
void init_array(uint32_t, uint32_t, uint32_t, double*, double*, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void kernel_gemm(uint32_t, uint32_t, uint32_t, double, double, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void print_array(uint32_t, uint32_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel0(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel1(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel2(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void kernel_gemm_OC_cudakernel0(uint64_t, uint64_t, uint64_t, double*, double) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void kernel_gemm_OC_cudakernel1(uint64_t, uint64_t, uint64_t, uint64_t, double*, double, uint64_t, double*, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));


/* Global Variable Definitions and Initialization */
uint8_t __FIXME_GLOBAL___OC_str[8] = { "%0.2lf " };
uint8_t __FIXME_GLOBAL___OC_str_OC_1[2] = { "\n" };


/* LLVM Intrinsic Builtin Function Bodies */
static __forceinline uint64_t llvm_add_u64(uint64_t a, uint64_t b) {
  uint64_t r = a + b;
  return r;
}
static __forceinline uint32_t llvm_mul_u32(uint32_t a, uint32_t b) {
  uint32_t r = a * b;
  return r;
}
static __forceinline uint64_t llvm_mul_u64(uint64_t a, uint64_t b) {
  uint64_t r = a * b;
  return r;
}
static __forceinline uint32_t llvm_srem_u32(int32_t a, int32_t b) {
  uint32_t r = a % b;
  return r;
}


/* Function Bodies */

int main(int argc, char ** argv) {
  int32_t dump_code;
  int32_t ni;
  int32_t nj;
  int32_t nk;
  uint8_t* alpha;
  uint8_t* beta;
  uint8_t* A;
  uint8_t* B;
  uint8_t* C;

  dump_code = atoi(argv[1]);
  ni = atoi(argv[2]);
  nj = atoi(argv[3]);
  nk = atoi(argv[4]);
  alpha = malloc(8);
  beta = malloc(8);
  A = malloc(ni * nk * 8);
  B = malloc(nk * nj * 8);
  C = malloc(ni * nj * 8);
  init_array(ni, nj, nk, ((double*)alpha), ((double*)beta), ((double*)C), ((double*)A), ((double*)B));
;
  kernel_gemm(ni, nj, nk, *((double*)alpha), *((double*)beta), ((double*)C), ((double*)A), ((double*)B));
;
  if (dump_code == 1) {
print_array(ni, nj, ((double*)C));
  }
free(((uint8_t*)((double*)C)));
free(((uint8_t*)((double*)A)));
free(((uint8_t*)((double*)B)));
  return 0;
}


void init_array(uint32_t ni, uint32_t nj, uint32_t nk, double* alpha, double* beta, double* C, double* A, double* B) {
  uint64_t __FIXME__size_C;    /* Address-exposed local */
  uint64_t __FIXME__size_A;    /* Address-exposed local */
  uint64_t __FIXME__size_B;    /* Address-exposed local */
  int64_t i;
  int64_t j;
unsigned threadsPerBlock = 256;
dim3 block0(threadsPerBlock, 1, 1);
dim3 grid0((ni+block0.x-1)/block0.x, 1, 1);

dim3 block2(threadsPerBlock, 1, 1);
dim3 grid2((nk+block2.x-1)/block2.x, 1, 1);

dim3 block1(threadsPerBlock, 1, 1);
dim3 grid1((ni+block1.x-1)/block1.x, 1, 1);


  *alpha = 32412;
  *beta = 2123;
  __FIXME__size_C = 0;
  __FIXME__size_A = 0;
  __FIXME__size_B = 0;
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__1 = i * nj + j;
  if (__FIXME__size_C < __FIXME__1) {
  __FIXME__size_C = __FIXME__1;
  }
}
}
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nk;   j = j + 1){
  uint64_t __FIXME__2 = i * nk + j;
  if (__FIXME__size_A < __FIXME__2) {
  __FIXME__size_A = __FIXME__2;
  }
}
}
for(int64_t i = 0; i < nk;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__3 = i * nj + j;
  if (__FIXME__size_B < __FIXME__3) {
  __FIXME__size_B = __FIXME__3;
  }
}
}
double* dev_A;
double* dev_B;
double* dev_C;

hipMalloc(&dev_A, size_A*sizeof(double));
hipMalloc(&dev_B, size_B*sizeof(double));
hipMalloc(&dev_C, size_C*sizeof(double));

hipMemcpy(dev_A, A, size_A*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_B, B, size_B*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_C, C, size_C*sizeof(double), hipMemcpyHostToDevice);

init_array_OC_cudakernel0<<<grid0, block0>>>(ni, nj, ni, nj, C);
init_array_OC_cudakernel1<<<grid1, block1>>>(ni, nk, ni, nk, A);
init_array_OC_cudakernel2<<<grid2, block2>>>(nk, nj, ni, nj, B);

hipMemcpy(A, dev_A, size_A*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(B, dev_B, size_B*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(C, dev_C, size_C*sizeof(double), hipMemcpyDeviceToHost);
  return;
}


void kernel_gemm(uint32_t ni, uint32_t nj, uint32_t nk, double alpha, double beta, double* C, double* A, double* B) {
  uint64_t __FIXME__size_C;    /* Address-exposed local */
  uint64_t __FIXME__size_A;    /* Address-exposed local */
  uint64_t __FIXME__size_B;    /* Address-exposed local */
  int64_t i;
  int64_t j;
  int64_t k;
unsigned threadsPerBlock = 256;
dim3 block0(threadsPerBlock, 1, 1);
dim3 grid0((ni+block0.x-1)/block0.x, 1, 1);

dim3 block1(threadsPerBlock, 1, 1);
dim3 grid1((ni+block1.x-1)/block1.x, 1, 1);


  __FIXME__size_C = 0;
  __FIXME__size_A = 0;
  __FIXME__size_B = 0;
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__4 = i * nj + j;
  if (__FIXME__size_C < __FIXME__4) {
  __FIXME__size_C = __FIXME__4;
  }
}
}
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
for(int64_t k = 0; k < nk;   k = k + 1){
  uint64_t __FIXME__5 = i * nk + k;
  if (__FIXME__size_A < __FIXME__5) {
  __FIXME__size_A = __FIXME__5;
  }
  uint64_t __FIXME__6 = k * nj + j;
  if (__FIXME__size_B < __FIXME__6) {
  __FIXME__size_B = __FIXME__6;
  }
  uint64_t __FIXME__7 = i * nj + j;
  if (__FIXME__size_C < __FIXME__7) {
  __FIXME__size_C = __FIXME__7;
  }
}
}
}
double* dev_A;
double* dev_B;
double* dev_C;

hipMalloc(&dev_A, size_A*sizeof(double));
hipMalloc(&dev_B, size_B*sizeof(double));
hipMalloc(&dev_C, size_C*sizeof(double));

hipMemcpy(dev_A, A, size_A*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_B, B, size_B*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_C, C, size_C*sizeof(double), hipMemcpyHostToDevice);

kernel_gemm_OC_cudakernel0<<<grid0, block0>>>(ni, nj, nj, C, beta);
kernel_gemm_OC_cudakernel1<<<grid1, block1>>>(ni, nj, nk, nk, A, alpha, nj, B, nj, C);

hipMemcpy(C, dev_C, size_C*sizeof(double), hipMemcpyDeviceToHost);
  return;
}


void print_array(uint32_t ni, uint32_t nj, double* C) {
  int64_t i;
  int64_t j;
  int32_t __FIXME__call11;

for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint32_t __FIXME__call = fprintf(stderr, __FIXME_GLOBAL___OC_str, C[(i * nj + j)]);
  if ((int)(i * ni + j) % (int)20 == 0) {
  fprintf(stderr, __FIXME_GLOBAL___OC_str_OC_1);
  }
}
}
  fprintf(stderr, __FIXME_GLOBAL___OC_str_OC_1);
}


__global__ void init_array_OC_cudakernel0(uint64_t __FIXME__8, uint64_t __FIXME__9, uint32_t __FIXME__ni, uint64_t __FIXME__10, double* __FIXME__C) {
  int64_t __FIXME__indvars_2e_iv14;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next15;

  __FIXME__indvars_2e_iv14 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv14 < __FIXME__8) {
for(int64_t i = 0; i < __FIXME__9;   i = i + 1){
  __FIXME__C[(__FIXME__indvars_2e_iv14 * __FIXME__10 + i)] = (((double)(__FIXME__indvars_2e_iv14) * (double)(i)) / (double)(__FIXME__ni));
}
  __FIXME__indvars_2e_iv14 + 1;
  }
  return;
}


__global__ void init_array_OC_cudakernel1(uint64_t __FIXME__11, uint64_t __FIXME__12, uint32_t __FIXME__ni, uint64_t __FIXME__13, double* __FIXME__A) {
  int64_t __FIXME__indvars_2e_iv8;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next9;

  __FIXME__indvars_2e_iv8 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv8 < __FIXME__11) {
for(int64_t i = 0; i < __FIXME__12;   i = i + 1){
  __FIXME__A[(__FIXME__indvars_2e_iv8 * __FIXME__13 + i)] = (((double)(__FIXME__indvars_2e_iv8) * (double)(i)) / (double)(__FIXME__ni));
}
  __FIXME__indvars_2e_iv8 + 1;
  }
  return;
}


__global__ void init_array_OC_cudakernel2(uint64_t __FIXME__14, uint64_t __FIXME__15, uint32_t __FIXME__ni, uint64_t __FIXME__16, double* __FIXME__B) {
  int64_t __FIXME__indvars_2e_iv2;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next3;

  __FIXME__indvars_2e_iv2 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv2 < __FIXME__14) {
for(int64_t i = 0; i < __FIXME__15;   i = i + 1){
  __FIXME__B[(__FIXME__indvars_2e_iv2 * __FIXME__16 + i)] = (((double)(__FIXME__indvars_2e_iv2) * (double)(i)) / (double)(__FIXME__ni));
}
  __FIXME__indvars_2e_iv2 + 1;
  }
  return;
}


__global__ void kernel_gemm_OC_cudakernel0(uint64_t __FIXME__17, uint64_t __FIXME__18, uint64_t __FIXME__19, double* __FIXME__C, double __FIXME__beta) {
  int64_t __FIXME__indvars_2e_iv14;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next15;

  __FIXME__indvars_2e_iv14 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv14 < __FIXME__17) {
for(int64_t i = 0; i < __FIXME__18;   i = i + 1){
  __FIXME__C[(__FIXME__indvars_2e_iv14 * __FIXME__19 + i)] = (__FIXME__C[(__FIXME__indvars_2e_iv14 * __FIXME__19 + i)] * __FIXME__beta);
}
  __FIXME__indvars_2e_iv14 + 1;
  }
  return;
}


__global__ void kernel_gemm_OC_cudakernel1(uint64_t __FIXME__20, uint64_t __FIXME__21, uint64_t __FIXME__22, uint64_t __FIXME__23, double* __FIXME__A, double __FIXME__alpha, uint64_t __FIXME__24, double* __FIXME__B, uint64_t __FIXME__25, double* __FIXME__C) {
  int64_t __FIXME__indvars_2e_iv7;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next8;
  int64_t j;

  __FIXME__indvars_2e_iv7 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv7 < __FIXME__20) {
for(int64_t i = 0; i < __FIXME__21;   i = i + 1){
for(int64_t j = 0; j < __FIXME__22;   j = j + 1){
  __FIXME__C[(__FIXME__indvars_2e_iv7 * __FIXME__25 + i)] = (__FIXME__C[(__FIXME__indvars_2e_iv7 * __FIXME__25 + i)] + ((__FIXME__alpha * __FIXME__A[(__FIXME__indvars_2e_iv7 * __FIXME__23 + j)]) * __FIXME__B[(j * __FIXME__24 + i)]));
}
}
  __FIXME__indvars_2e_iv7 + 1;
  }
  return;
}

