/**
 * fdtd-2d.c: This file is part of the PolyBench 3.0 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

#define TMAX 100
#define NX 4000
#define NY 4000

static unsigned num_blocks(int num, int factor) {
  return (num + factor - 1) / factor;
}


__global__ void kernel_splat(int tmax,
                             int nx,
                             int ny,
                             double *ex, double *ey, double *hz, double fict[], int t) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if (j < ny)
    ey[0 * ny + j] = fict[t];
}


__global__ void kernel_ey(int tmax,
                          int nx,
                          int ny,
                          double *ex, double *ey, double *hz, double fict[], int t) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if (i < nx && j < ny)
    ey[i * ny + j] -= (double)(0.5) * (hz[i * ny + j] - hz[(i - 1) * ny + j]);
}



__global__ void kernel_ex(int tmax,
                          int nx,
                          int ny,
                          double *ex, double *ey, double *hz, double fict[], int t) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + 1;

  if (i < nx && j < ny)
    ex[i * ny + j] -= (double)(0.5) * (hz[i * ny + j] - hz[i * ny + j - 1]);
}



__global__ void kernel_hz(int tmax,
                          int nx,
                          int ny,
                          double *ex, double *ey, double *hz, double fict[], int t) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;

  if (i < nx - 1 && j < ny - 1)
    hz[i * ny + j] -= (double)(0.7) * (ex[i * ny + j + 1] - ex[i * ny + j] + ey[(i + 1) * ny + j] - ey[i * ny + j]);
}



static void kernel(int tmax,
                   int nx,
                   int ny,
                   double *ex, double *ey, double *hz, double *fict) {
  const unsigned threadsPerBlock = 256;



  for (int t = 0; t < tmax; t++) {
    kernel_splat<<<threadsPerBlock, num_blocks(ny, threadsPerBlock)>>>(tmax, nx, ny, ex, ey, hz, fict, t);

    {
      dim3 block(threadsPerBlock / 32, 32, 1);
      dim3 grid(num_blocks(nx - 1, block.x), num_blocks(ny, block.y), 1);
      kernel_ey<<<grid, block>>>(tmax, nx, ny, ex, ey, hz, fict, t);
    }


    {
      dim3 block(threadsPerBlock / 32, 32, 1);
      dim3 grid(num_blocks(nx, block.x), num_blocks(ny - 1, block.y), 1);
      kernel_ex<<<grid, block>>>(tmax, nx, ny, ex, ey, hz, fict, t);
    }

    {
      dim3 block(threadsPerBlock / 32, 32, 1);
      dim3 grid(num_blocks(nx - 1, block.x), num_blocks(ny - 1, block.y), 1);
      kernel_hz<<<grid, block>>>(tmax, nx, ny, ex, ey, hz, fict, t);
    }
  }

}

/* Array initialization. */
static
void init_array (int nx,
		 int ny,
		 double *ex,
		 double *ey,
		 double *hz,
		 double *_fict_)
{
  int i, j;

  for (i = 0; i < ny; i++)
    _fict_[i] = (double) i;
  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
    {
	    ex[i*ny+j] = ((double) i*(j+1)) / nx;
	    ey[i*ny+j] = ((double) i*(j+2)) / ny;
	    hz[i*ny+j] = ((double) i*(j+3)) / nx;
      }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx,
		 int ny,
		 double *ex,
		 double *ey,
		 double *hz)
{
  int i, j;

  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
      fprintf(stderr, "%0.2lf ", ex[i*ny+j]);
      fprintf(stderr, "%0.2lf ", ey[i*ny+j]);
      fprintf(stderr, "%0.2lf ", hz[i*ny+j]);
      if ((i * nx + j) % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_fdtd_2d(int tmax,
		    int nx,
		    int ny,
		    double ex[nx][ny],
		    double ey[nx][ny],
		    double hz[nx][ny],
		    double _fict_[ny])
{
  int t, i, j;

  for(t = 0; t < tmax; t++)
    {
      for (j = 0; j < ny; j++)
	      ey[0][j] = _fict_[t];
      for (i = 1; i < nx; i++)
	      for (j = 0; j < ny; j++)
	        ey[i][j] = ey[i][j] - 0.5*(hz[i][j]-hz[i-1][j]);
      for (i = 0; i < nx; i++)
	      for (j = 1; j < ny; j++)
	        ex[i][j] = ex[i][j] - 0.5*(hz[i][j]-hz[i][j-1]);
      for (i = 0; i < nx - 1; i++)
	      for (j = 0; j < ny - 1; j++)
	        hz[i][j] = hz[i][j] - 0.7*  (ex[i][j+1] - ex[i][j] +
				       ey[i+1][j] - ey[i][j]);
    }
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int tmax = atoi(argv[2]);
  int nx = atoi(argv[3]);
  int ny = atoi(argv[4]);
  int dump_code = atoi(argv[1]);

  /* Variable declaration/allocation. */
  double *ex = (double*)malloc(nx*ny*sizeof(double));
  double *ey = (double*)malloc(nx*ny*sizeof(double));
  double *hz = (double*)malloc(nx*ny*sizeof(double));
  double *_fict_ = (double*)malloc(ny*sizeof(double));

  /* Initialize array(s). */
  init_array (nx, ny,
	      ex,
	      ey,
	      hz,
	      _fict_);
  double *dev_ex;
  double *dev_ey;
  double *dev_hz;
  double *dev_fict;
  hipMalloc(&dev_ex, nx*ny*sizeof(double));
  hipMalloc(&dev_ey, nx*ny*sizeof(double));
  hipMalloc(&dev_hz, nx*ny*sizeof(double));
  hipMalloc(&dev_fict, ny*sizeof(double));
  hipMemcpy(dev_ex, ex, nx*ny*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_ey, ey, nx*ny*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_hz, hz, nx*ny*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_fict, _fict_, ny*sizeof(double), hipMemcpyHostToDevice);
  /* Run kernel. */
  kernel(tmax, nx, ny, dev_ex, dev_ey, dev_hz, dev_fict);
  hipMemcpy(ex, dev_ex, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(ey, dev_ey, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(hz, dev_hz, nx*ny*sizeof(double), hipMemcpyDeviceToHost);


  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  if(dump_code == 1) print_array(nx, ny, ex, ey, hz);

  /* Be clean. */
  free((void*)ex);
  free((void*)ey);
  free((void*)hz);
  free((void*)_fict_);

  return 0;
}
