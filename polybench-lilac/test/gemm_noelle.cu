#include "hip/hip_runtime.h"
/* Provide Declarations */
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#ifndef __cplusplus
typedef unsigned char bool;
#endif

#ifndef _MSC_VER
#define __forceinline __attribute__((always_inline)) inline
#endif

#if defined(__GNUC__)
#define  __ATTRIBUTELIST__(x) __attribute__(x)
#else
#define  __ATTRIBUTELIST__(x)  
#endif

#ifdef _MSC_VER  /* Can only support "linkonce" vars with GCC */
#define __attribute__(X)
#endif



/* Global Declarations */

/* Types Declarations */
struct __FIXME__l_struct_struct_OC__IO_FILE;

/* Function definitions */

/* Types Definitions */
struct __FIXME__l_array_1_uint8_t {
  uint8_t array[1];
};
struct __FIXME__l_array_20_uint8_t {
  uint8_t array[20];
};
struct __FIXME__l_struct_struct_OC__IO_FILE {
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field0;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field1;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field2;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field3;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field4;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field5;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field6;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field7;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field8;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field9;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field10;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field11;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field12;
  struct __FIXME__l_struct_struct_OC__IO_FILE* __FIXME__l_struct_struct_OC__IO_FILE_field13;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field14;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field15;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field16;
  uint16_t __FIXME__l_struct_struct_OC__IO_FILE_field17;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field18;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field19[1];
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field20;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field21;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field22;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field23;
  struct __FIXME__l_struct_struct_OC__IO_FILE* __FIXME__l_struct_struct_OC__IO_FILE_field24;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field25;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field26;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field27;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field28[20];
};

/* External Global Variable Declarations */

/* Function Declarations */
int main(int, char **) __ATTRIBUTELIST__((noinline, nothrow));
void init_array(uint32_t, uint32_t, uint32_t, double*, double*, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void print_array(uint32_t, uint32_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel0(uint64_t, uint32_t, uint32_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel1(uint64_t, uint32_t, uint32_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel2(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));


/* Global Variable Definitions and Initialization */
uint8_t __FIXME_GLOBAL___OC_str[8] = { "%0.2lf " };
uint8_t __FIXME_GLOBAL___OC_str_OC_1[2] = { "\n" };


/* LLVM Intrinsic Builtin Function Bodies */
static __forceinline uint32_t llvm_add_u32(uint32_t a, uint32_t b) {
  uint32_t r = a + b;
  return r;
}
static __forceinline uint64_t llvm_add_u64(uint64_t a, uint64_t b) {
  uint64_t r = a + b;
  return r;
}
static __forceinline uint32_t llvm_mul_u32(uint32_t a, uint32_t b) {
  uint32_t r = a * b;
  return r;
}
static __forceinline uint64_t llvm_mul_u64(uint64_t a, uint64_t b) {
  uint64_t r = a * b;
  return r;
}
static __forceinline uint32_t llvm_srem_u32(int32_t a, int32_t b) {
  uint32_t r = a % b;
  return r;
}


/* Function Bodies */

int main(int argc, char ** argv) {
  int32_t ni;
  int32_t nj;
  int32_t nk;
  uint8_t* alpha;
  uint8_t* beta;
  uint8_t* A;
  uint8_t* B;
  uint8_t* C;

  ni = atoi(argv[2]);
  nj = atoi(argv[3]);
  nk = atoi(argv[4]);
  alpha = malloc(8);
  beta = malloc(8);
  A = malloc(ni * nk * 8);
  B = malloc(nk * nj * 8);
  C = malloc(ni * nj * 8);
init_array(ni, nj, nk, ((double*)alpha), ((double*)beta), ((double*)C), ((double*)A), ((double*)B));
print_array(ni, nj, ((double*)C));
free(((uint8_t*)((double*)C)));
free(((uint8_t*)((double*)A)));
free(((uint8_t*)((double*)B)));
  return 0;
}


void init_array(uint32_t ni, uint32_t nj, uint32_t nk, double* alpha, double* beta, double* C, double* A, double* B) {
  uint64_t __FIXME__size_C;    /* Address-exposed local */
  uint64_t __FIXME__size_A;    /* Address-exposed local */
  uint64_t __FIXME__size_B;    /* Address-exposed local */
  int32_t i;
  int64_t j;
unsigned threadsPerBlock = 256;
dim3 block2(threadsPerBlock, 1, 1);
dim3 grid2((nk+block2.x-1)/block2.x, 1, 1);

dim3 block0(threadsPerBlock, 1, 1);
dim3 grid0((nj+block0.x-1)/block0.x, 1, 1);

dim3 block1(threadsPerBlock, 1, 1);
dim3 grid1((ni+block1.x-1)/block1.x, 1, 1);


  *alpha = 32412;
  *beta = 2123;
  __FIXME__size_C = 0;
  __FIXME__size_A = 0;
  __FIXME__size_B = 0;
for(int32_t i = 0; i < ni;   i = i + 1){
#pragma omp parallel for 
for(int64_t j = 0; j < nj;   j = j + 1){
  if (__FIXME__size_C < j) {
  __FIXME__size_C = j;
  }
}
}
for(int64_t i = 0; i < ni;   i = i + 1){
for(int32_t j = 0; j < nk;   j = j + 1){
  if (__FIXME__size_A < i) {
  __FIXME__size_A = i;
  }
}
}
for(int64_t i = 0; i < nk;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__1 = i * nj + j;
  if (__FIXME__size_B < __FIXME__1) {
  __FIXME__size_B = __FIXME__1;
  }
}
}
for(int32_t i = 0; i < ni;   i = i + 1){
double* dev_A;
double* dev_B;
double* dev_C;

hipMalloc(&dev_A, size_A*sizeof(double));
hipMalloc(&dev_B, size_B*sizeof(double));
hipMalloc(&dev_C, size_C*sizeof(double));

hipMemcpy(dev_A, A, size_A*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_B, B, size_B*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dev_C, C, size_C*sizeof(double), hipMemcpyHostToDevice);

init_array_OC_cudakernel0<<<grid0, block0>>>(nj, i, ni, C);
}
init_array_OC_cudakernel1<<<grid1, block1>>>(ni, nk, ni, A);
init_array_OC_cudakernel2<<<grid2, block2>>>(nk, nj, ni, nj, B);

hipMemcpy(A, dev_A, size_A*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(B, dev_B, size_B*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(C, dev_C, size_C*sizeof(double), hipMemcpyDeviceToHost);
  return;
}


void print_array(uint32_t ni, uint32_t nj, double* C) {
  int64_t i;
  int64_t j;
  int32_t __FIXME__call11;

for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint32_t __FIXME__call = fprintf(stderr, __FIXME_GLOBAL___OC_str, C[(i * nj + j)]);
  if ((int)(i * ni + j) % (int)20 == 0) {
  fprintf(stderr, __FIXME_GLOBAL___OC_str_OC_1);
  }
}
}
  fprintf(stderr, __FIXME_GLOBAL___OC_str_OC_1);
}


__global__ void init_array_OC_cudakernel0(uint64_t __FIXME__2, uint32_t __FIXME__i_2e_0, uint32_t __FIXME__ni, double* __FIXME__C) {
  int64_t __FIXME__indvars_2e_iv7;
  uint64_t __FIXME__indvars_2e_iv_2e_next8;

  __FIXME__indvars_2e_iv7 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv7 < __FIXME__2) {
  __FIXME__C[__FIXME__indvars_2e_iv7] = (((double)(__FIXME__i_2e_0) * (double)(__FIXME__indvars_2e_iv7)) / (double)(__FIXME__ni));
  __FIXME__indvars_2e_iv7 + 1;
  }
  return;
}


__global__ void init_array_OC_cudakernel1(uint64_t __FIXME__3, uint32_t __FIXME__nk, uint32_t __FIXME__ni, double* __FIXME__A) {
  int64_t __FIXME__indvars_2e_iv5;
  int32_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next6;

  __FIXME__indvars_2e_iv5 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv5 < __FIXME__3) {
for(int32_t i = 0; i < __FIXME__nk;   i = i + 1){
  __FIXME__A[__FIXME__indvars_2e_iv5] = (((double)(__FIXME__indvars_2e_iv5) * (double)(i)) / (double)(__FIXME__ni));
}
  __FIXME__indvars_2e_iv5 + 1;
  }
  return;
}


__global__ void init_array_OC_cudakernel2(uint64_t __FIXME__4, uint64_t __FIXME__5, uint32_t __FIXME__ni, uint64_t __FIXME__6, double* __FIXME__B) {
  int64_t __FIXME__indvars_2e_iv2;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next3;

  __FIXME__indvars_2e_iv2 = blockDim.x * blockIdx.x + threadIdx.x + 0;
  if (__FIXME__indvars_2e_iv2 < __FIXME__4) {
for(int64_t i = 0; i < __FIXME__5;   i = i + 1){
  __FIXME__B[(__FIXME__indvars_2e_iv2 * __FIXME__6 + i)] = (((double)(__FIXME__indvars_2e_iv2) * (double)(i)) / (double)(__FIXME__ni));
}
  __FIXME__indvars_2e_iv2 + 1;
  }
  return;
}

