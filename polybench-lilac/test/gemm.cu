/**
 * gemm.c: This file is part of the PolyBench/C 3.2 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

  static
void init_array(int ni, int nj, int nk,
    double *C,
    double *A,
    double *B)
{
  int i, j;


  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++)
      C[i*nj+j] = ((double) i*j) / ni;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nk; j++)
      A[i*nk+j] = ((double) i*j) / ni;
  for (i = 0; i < nk; i++)
    for (j = 0; j < nj; j++)
      B[i*nj+j] = ((double) i*j) / ni;
}




  static
void print_array(int ni, int nj,
    double *C)
{
  int i = 0, j = 0;

//  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
      fprintf (stderr, "%0.2lf", C[i*nj+j]);
      if ((i * ni + j) % 20 == 0) {
        if(i == 0) fprintf(stderr, "\n");
        fprintf (stderr, "\n");
      }
      else fprintf(stderr, " ");
    }
}


int main(int argc, char** argv)
{

  int dump_code = atoi(argv[1]);
  int ni = atoi(argv[2]);
  int nj = atoi(argv[3]);
  int nk = atoi(argv[4]);

  double *A = (double*)malloc(ni*nk*sizeof(double));
  double *B = (double*)malloc(nk*nj*sizeof(double));
  double *C = (double*)malloc(ni*nj*sizeof(double));

  double alpha = 32412;
  double beta = 2123;

  init_array (ni, nj, nk,
      C,
      A,
      B);

  if (dump_code == 1) print_array(ni, nj, C);


  free((void*)C);
  free((void*)A);
  free((void*)B);
  return 0;
}
