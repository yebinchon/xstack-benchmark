/* Provide Declarations */

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#ifndef __cplusplus
typedef unsigned char bool;
#endif

#ifndef _MSC_VER
#define __forceinline __attribute__((always_inline)) inline
#endif

#if defined(__GNUC__)
#define  __ATTRIBUTELIST__(x) __attribute__(x)
#else
#define  __ATTRIBUTELIST__(x)  
#endif

#ifdef _MSC_VER  /* Can only support "linkonce" vars with GCC */
#define __attribute__(X)
#endif



/* Global Declarations */

/* Types Declarations */
struct __FIXME__l_struct_struct_OC__IO_FILE;

/* Function definitions */

/* Types Definitions */
struct __FIXME__l_array_1_uint8_t {
  uint8_t array[1];
};
struct __FIXME__l_array_20_uint8_t {
  uint8_t array[20];
};
struct __FIXME__l_struct_struct_OC__IO_FILE {
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field0;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field1;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field2;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field3;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field4;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field5;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field6;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field7;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field8;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field9;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field10;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field11;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field12;
  struct __FIXME__l_struct_struct_OC__IO_FILE* __FIXME__l_struct_struct_OC__IO_FILE_field13;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field14;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field15;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field16;
  uint16_t __FIXME__l_struct_struct_OC__IO_FILE_field17;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field18;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field19[1];
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field20;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field21;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field22;
  void* __FIXME__l_struct_struct_OC__IO_FILE_field23;
  struct __FIXME__l_struct_struct_OC__IO_FILE* __FIXME__l_struct_struct_OC__IO_FILE_field24;
  uint8_t* __FIXME__l_struct_struct_OC__IO_FILE_field25;
  uint64_t __FIXME__l_struct_struct_OC__IO_FILE_field26;
  uint32_t __FIXME__l_struct_struct_OC__IO_FILE_field27;
  uint8_t __FIXME__l_struct_struct_OC__IO_FILE_field28[20];
};

/* External Global Variable Declarations */

/* Function Declarations */
int main(int, char **) __ATTRIBUTELIST__((noinline, nothrow));
void init_array(uint32_t, uint32_t, uint32_t, uint32_t, double*, double*, double*, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void kernel_2mm(uint32_t, uint32_t, uint32_t, uint32_t, double, double, double*, double*, double*, double*, double*) __ATTRIBUTELIST__((noinline, nothrow));
void print_array(uint32_t, uint32_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel0(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel1(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel2(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void init_array_OC_cudakernel3(uint64_t, uint64_t, uint32_t, uint64_t, double*) __ATTRIBUTELIST__((noinline, nothrow));
__global__ void kernel_2mm_OC_cudakernel0(uint64_t, uint64_t, uint64_t, double*, uint64_t, uint64_t, double*, double, uint64_t, double*, uint64_t) __ATTRIBUTELIST__((noinline, nothrow));


/* Global Variable Definitions and Initialization */
uint8_t __FIXME_GLOBAL___OC_str[8] = { "%0.2lf " };
uint8_t __FIXME_GLOBAL___OC_str_OC_1[2] = { "\n" };


/* LLVM Intrinsic Builtin Function Bodies */
static __forceinline uint64_t llvm_add_u64(uint64_t a, uint64_t b) {
  uint64_t r = a + b;
  return r;
}
static __forceinline uint32_t llvm_mul_u32(uint32_t a, uint32_t b) {
  uint32_t r = a * b;
  return r;
}
static __forceinline uint64_t llvm_mul_u64(uint64_t a, uint64_t b) {
  uint64_t r = a * b;
  return r;
}
static __forceinline uint32_t llvm_srem_u32(int32_t a, int32_t b) {
  uint32_t r = a % b;
  return r;
}


/* Function Bodies */

int main(int argc, char ** argv) {
  double alpha;    /* Address-exposed local */
  double beta;    /* Address-exposed local */
  int32_t dump_code;
  int32_t ni;
  int32_t nj;
  int32_t nk;
  int32_t nl;
  uint8_t* tmp;
  uint8_t* A;
  uint8_t* B;
  uint8_t* C;
  uint8_t* D;

//INSERT COMMENT IFELSE: entry
  dump_code = atoi(argv[1]);
  ni = atoi(argv[2]);
  nj = atoi(argv[3]);
  nk = atoi(argv[4]);
  nl = atoi(argv[5]);
  tmp = (uint8_t*)malloc(ni * nj * 8);
  A = (uint8_t*)malloc(ni * nk * 8);
  B = (uint8_t*)malloc(nk * nj * 8);
  C = (uint8_t*)malloc(nl * nj * 8);
  D = (uint8_t*)malloc(ni * nl * 8);
  init_array(ni, nj, nk, nl, (&alpha), (&beta), ((double*)A), ((double*)B), ((double*)C), ((double*)D));
;
  kernel_2mm(ni, nj, nk, nl, alpha, beta, ((double*)tmp), ((double*)A), ((double*)B), ((double*)C), ((double*)D));
;
  if (dump_code == 1) {
print_array(ni, nl, ((double*)D));
  }
free(((uint8_t*)((double*)tmp)));
free(((uint8_t*)((double*)A)));
free(((uint8_t*)((double*)B)));
free(((uint8_t*)((double*)C)));
free(((uint8_t*)((double*)D)));
  return 0;
}


//INSERT COMMENT FUNCTION: init_array
void init_array(uint32_t ni, uint32_t nj, uint32_t nk, uint32_t nl, double* alpha, double* beta, double* A, double* B, double* C, double* D) {
  uint64_t __FIXME__size_A;    /* Address-exposed local */
  uint64_t __FIXME__size_B;    /* Address-exposed local */
  uint64_t __FIXME__size_C;    /* Address-exposed local */
  uint64_t __FIXME__size_D;    /* Address-exposed local */
  int64_t i;
  int64_t j;
unsigned threadsPerBlock = 256;
dim3 block0(threadsPerBlock, 1, 1);
dim3 grid0((ni+block0.x-1)/block0.x, 1, 1);

dim3 block3(threadsPerBlock, 1, 1);
dim3 grid3((ni+block3.x-1)/block3.x, 1, 1);

dim3 block2(threadsPerBlock, 1, 1);
dim3 grid2((nl+block2.x-1)/block2.x, 1, 1);

dim3 block1(threadsPerBlock, 1, 1);
dim3 grid1((nk+block1.x-1)/block1.x, 1, 1);


  *alpha = 32412;
  *beta = 2123;
  __FIXME__size_A = 0;
  __FIXME__size_B = 0;
  __FIXME__size_C = 0;
  __FIXME__size_D = 0;
//INSERT COMMENT LOOP: for.cond.bound
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nk;   j = j + 1){
  uint64_t __FIXME__1 = i * nk + j;
  if (__FIXME__size_A < __FIXME__1) {
  __FIXME__size_A = __FIXME__1;
  }
}
}
//INSERT COMMENT LOOP: for.cond10.bound
for(int64_t i = 0; i < nk;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__2 = i * nj + j;
  if (__FIXME__size_B < __FIXME__2) {
  __FIXME__size_B = __FIXME__2;
  }
}
}
//INSERT COMMENT LOOP: for.cond34.bound
for(int64_t i = 0; i < nl;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__3 = i * nj + j;
  if (__FIXME__size_C < __FIXME__3) {
  __FIXME__size_C = __FIXME__3;
  }
}
}
//INSERT COMMENT LOOP: for.cond58.bound
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nl;   j = j + 1){
  uint64_t __FIXME__4 = i * nl + j;
  if (__FIXME__size_D < __FIXME__4) {
  __FIXME__size_D = __FIXME__4;
  }
}
}
double* __FIXME__dev_A;
double* __FIXME__dev_B;
double* __FIXME__dev_C;
double* __FIXME__dev_D;

hipMalloc(&__FIXME__dev_A, __FIXME__size_A*sizeof(double));
hipMalloc(&__FIXME__dev_B, __FIXME__size_B*sizeof(double));
hipMalloc(&__FIXME__dev_C, __FIXME__size_C*sizeof(double));
hipMalloc(&__FIXME__dev_D, __FIXME__size_D*sizeof(double));

hipMemcpy(__FIXME__dev_A, A, __FIXME__size_A*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(__FIXME__dev_B, B, __FIXME__size_B*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(__FIXME__dev_C, C, __FIXME__size_C*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(__FIXME__dev_D, D, __FIXME__size_D*sizeof(double), hipMemcpyHostToDevice);

init_array_OC_cudakernel0<<<grid0, block0>>>(ni, nk, ni, nk, A);
init_array_OC_cudakernel1<<<grid1, block1>>>(nk, nj, nj, nj, B);
init_array_OC_cudakernel2<<<grid2, block2>>>(nl, nj, nl, nj, C);
init_array_OC_cudakernel3<<<grid3, block3>>>(ni, nl, nk, nl, D);

hipMemcpy(A, __FIXME__dev_A, __FIXME__size_A*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(B, __FIXME__dev_B, __FIXME__size_B*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(C, __FIXME__dev_C, __FIXME__size_C*sizeof(double), hipMemcpyDeviceToHost);
hipMemcpy(D, __FIXME__dev_D, __FIXME__size_D*sizeof(double), hipMemcpyDeviceToHost);
  return;
}


//INSERT COMMENT FUNCTION: kernel_2mm
void kernel_2mm(uint32_t ni, uint32_t nj, uint32_t nk, uint32_t nl, double alpha, double beta, double* tmp, double* A, double* B, double* C, double* D) {
  uint64_t __FIXME__size_tmp;    /* Address-exposed local */
  uint64_t __FIXME__size_A;    /* Address-exposed local */
  uint64_t __FIXME__size_B;    /* Address-exposed local */
  int64_t i;
  int64_t j;
  int64_t k;
unsigned threadsPerBlock = 256;
dim3 block0(threadsPerBlock, 1, 1);
dim3 grid0((ni+block0.x-1)/block0.x, 1, 1);


  __FIXME__size_tmp = 0;
  __FIXME__size_A = 0;
  __FIXME__size_B = 0;
//INSERT COMMENT LOOP: for.cond.bound
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nj;   j = j + 1){
  uint64_t __FIXME__5 = i * nj + j;
  if (__FIXME__size_tmp < __FIXME__5) {
  __FIXME__size_tmp = __FIXME__5;
  }
for(int64_t k = 0; k < nk;   k = k + 1){
  uint64_t __FIXME__6 = i * nk + k;
  if (__FIXME__size_A < __FIXME__6) {
  __FIXME__size_A = __FIXME__6;
  }
  uint64_t __FIXME__7 = k * nj + j;
  if (__FIXME__size_B < __FIXME__7) {
  __FIXME__size_B = __FIXME__7;
  }
  uint64_t __FIXME__8 = i * nj + j;
  if (__FIXME__size_tmp < __FIXME__8) {
  __FIXME__size_tmp = __FIXME__8;
  }
}
}
}
double* __FIXME__dev_A;
double* __FIXME__dev_B;
double* __FIXME__dev_tmp;

hipMalloc(&__FIXME__dev_A, __FIXME__size_A*sizeof(double));
hipMalloc(&__FIXME__dev_B, __FIXME__size_B*sizeof(double));
hipMalloc(&__FIXME__dev_tmp, __FIXME__size_tmp*sizeof(double));

hipMemcpy(__FIXME__dev_A, A, __FIXME__size_A*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(__FIXME__dev_B, B, __FIXME__size_B*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(__FIXME__dev_tmp, tmp, __FIXME__size_tmp*sizeof(double), hipMemcpyHostToDevice);

kernel_2mm_OC_cudakernel0<<<grid0, block0>>>(ni, nj, nj, tmp, nk, nk, A, alpha, nj, B, nj);

hipMemcpy(tmp, __FIXME__dev_tmp, __FIXME__size_tmp*sizeof(double), hipMemcpyDeviceToHost);
////INSERT COMMENT LOOP: for.cond28
//for(int64_t i = 0; i < ni;   i = i + 1){
//for(int64_t j = 0; j < nl;   j = j + 1){
//  D[(i * nj + j)] = (D[(i * nj + j)] * beta);
//for(int64_t k = 0; k < nj;   k = k + 1){
//  D[(i * nl + j)] = (D[(i * nl + j)] + (tmp[(i * nj + k)] * C[(k * nl + j)]));
//}
//}
//}
  return;
}


//INSERT COMMENT FUNCTION: print_array
void print_array(uint32_t ni, uint32_t nl, double* D) {
  int64_t i;
  int64_t j;
  int32_t __FIXME__call11;

//INSERT COMMENT LOOP: for.cond
for(int64_t i = 0; i < ni;   i = i + 1){
for(int64_t j = 0; j < nl;   j = j + 1){
  uint32_t __FIXME__call = fprintf(stderr, (char*)__FIXME_GLOBAL___OC_str, D[(i * nl + j)]);
  if ((int)(i * ni + j) % (int)20 == 0) {
  fprintf(stderr, (char*)__FIXME_GLOBAL___OC_str_OC_1);
  }
}
}
  fprintf(stderr, (char*)__FIXME_GLOBAL___OC_str_OC_1);
}


//INSERT COMMENT FUNCTION: init_array.cudakernel0
__global__ void init_array_OC_cudakernel0(uint64_t __FIXME__9, uint64_t __FIXME__10, uint32_t __FIXME__ni, uint64_t __FIXME__11, double* __FIXME__A) {
  int64_t __FIXME__indvars_2e_iv22;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next23;

  __FIXME__indvars_2e_iv22 = blockDim.x * blockIdx.x + threadIdx.x + 0;
//INSERT COMMENT IFELSE: for.cond
  if (__FIXME__indvars_2e_iv22 < __FIXME__9) {
for(int64_t i = 0; i < __FIXME__10;   i = i + 1){
  __FIXME__A[(__FIXME__indvars_2e_iv22 * __FIXME__11 + i)] = (((double)(__FIXME__indvars_2e_iv22) * (double)(i)) / (double)(__FIXME__ni));
}
  __FIXME__indvars_2e_iv22 + 1;
  }
  return;
}


//INSERT COMMENT FUNCTION: init_array.cudakernel1
__global__ void init_array_OC_cudakernel1(uint64_t __FIXME__12, uint64_t __FIXME__13, uint32_t __FIXME__nj, uint64_t __FIXME__14, double* __FIXME__B) {
  int64_t __FIXME__indvars_2e_iv16;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next17;

  __FIXME__indvars_2e_iv16 = blockDim.x * blockIdx.x + threadIdx.x + 0;
//INSERT COMMENT IFELSE: for.cond10
  if (__FIXME__indvars_2e_iv16 < __FIXME__12) {
for(int64_t i = 0; i < __FIXME__13;   i = i + 1){
  __FIXME__B[(__FIXME__indvars_2e_iv16 * __FIXME__14 + i)] = (((double)(__FIXME__indvars_2e_iv16) * (double)(i)) / (double)(__FIXME__nj));
}
  __FIXME__indvars_2e_iv16 + 1;
  }
  return;
}


//INSERT COMMENT FUNCTION: init_array.cudakernel2
__global__ void init_array_OC_cudakernel2(uint64_t __FIXME__15, uint64_t __FIXME__16, uint32_t __FIXME__nl, uint64_t __FIXME__17, double* __FIXME__C) {
  int64_t __FIXME__indvars_2e_iv10;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next11;

  __FIXME__indvars_2e_iv10 = blockDim.x * blockIdx.x + threadIdx.x + 0;
//INSERT COMMENT IFELSE: for.cond34
  if (__FIXME__indvars_2e_iv10 < __FIXME__15) {
for(int64_t i = 0; i < __FIXME__16;   i = i + 1){
  __FIXME__C[(__FIXME__indvars_2e_iv10 * __FIXME__17 + i)] = (((double)(__FIXME__indvars_2e_iv10) * (double)((i + 3))) / (double)(__FIXME__nl));
}
  __FIXME__indvars_2e_iv10 + 1;
  }
  return;
}


//INSERT COMMENT FUNCTION: init_array.cudakernel3
__global__ void init_array_OC_cudakernel3(uint64_t __FIXME__18, uint64_t __FIXME__19, uint32_t __FIXME__nk, uint64_t __FIXME__20, double* __FIXME__D) {
  int64_t __FIXME__indvars_2e_iv3;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next4;

  __FIXME__indvars_2e_iv3 = blockDim.x * blockIdx.x + threadIdx.x + 0;
//INSERT COMMENT IFELSE: for.cond58
  if (__FIXME__indvars_2e_iv3 < __FIXME__18) {
for(int64_t i = 0; i < __FIXME__19;   i = i + 1){
  __FIXME__D[(__FIXME__indvars_2e_iv3 * __FIXME__20 + i)] = (((double)(__FIXME__indvars_2e_iv3) * (double)((i + 2))) / (double)(__FIXME__nk));
}
  __FIXME__indvars_2e_iv3 + 1;
  }
  return;
}


//INSERT COMMENT FUNCTION: kernel_2mm.cudakernel0
__global__ void kernel_2mm_OC_cudakernel0(uint64_t __FIXME__21, uint64_t __FIXME__22, uint64_t __FIXME__23, double* __FIXME__tmp, uint64_t __FIXME__24, uint64_t __FIXME__25, double* __FIXME__A, double __FIXME__alpha, uint64_t __FIXME__26, double* __FIXME__B, uint64_t __FIXME__27) {
  int64_t __FIXME__indvars_2e_iv22;
  int64_t i;
  uint64_t __FIXME__indvars_2e_iv_2e_next23;
  int64_t j;

  __FIXME__indvars_2e_iv22 = blockDim.x * blockIdx.x + threadIdx.x + 0;
//INSERT COMMENT IFELSE: for.cond
  if (__FIXME__indvars_2e_iv22 < __FIXME__21) {
for(int64_t i = 0; i < __FIXME__22;   i = i + 1){
  __FIXME__tmp[(__FIXME__indvars_2e_iv22 * __FIXME__23 + i)] = 0;
for(int64_t j = 0; j < __FIXME__24;   j = j + 1){
  __FIXME__tmp[(__FIXME__indvars_2e_iv22 * __FIXME__27 + i)] = (__FIXME__tmp[(__FIXME__indvars_2e_iv22 * __FIXME__27 + i)] + ((__FIXME__alpha * __FIXME__A[(__FIXME__indvars_2e_iv22 * __FIXME__25 + j)]) * __FIXME__B[(j * __FIXME__26 + i)]));
}
}
  __FIXME__indvars_2e_iv22 + 1;
  }
  return;
}

